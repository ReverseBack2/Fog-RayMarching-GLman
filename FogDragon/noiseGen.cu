#include <iostream>
#include <stdio.h>
#include <string>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <bits/stdc++.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


using namespace std;
///////////////////////////////////////////
// STRUCTS
///////////////////////////////////////////

//3D array of distances to points
typedef struct {
	int res;
	float*** field;
} DistanceField;

//3D array of int noise values
typedef struct {
	int res;
	int*** field;
} NoiseField;

//coord to keep track of array indexes
typedef struct {
	int x;
	int y;
	int z;
} coord;

//coord but in floats for points to calc
typedef struct {
	float* loc;
} point;

//3D array of points for noise field
typedef struct {
	int res;		// how many points in one direction
	float CS;		// side length of each point section
	point*** field;
} PointField;

//Point generation code
void printCoord(float x, float y, float z, float CS, PointField PF);
float genFloat( float max );
void genPoints ( PointField &PF );

//Noise generation code
int distVal( float dist );
__device__ void getPointfromCoord( coord ID, PointField PF, point &p );
__global__ void calcDistance( PointField PF, DistanceField* DF );
__global__ void calcNoise( DistanceField DF, NoiseField* NF );



///////////////////////////////////////////
// Main Program
///////////////////////////////////////////

int main() {
	cout << "starting\n";

	int resolution = 512;
	DistanceField* DF;
	NoiseField* NF;

	hipMallocManaged(&DF, sizeof(DistanceField));
	hipMallocManaged(&NF, sizeof(NoiseField));

	//allocate 3D field of distance and noise
	hipMallocManaged(&(DF->field), resolution*sizeof(float**));
	hipMallocManaged(&(NF->field), resolution*sizeof(int**));

	for (int i = 0; i < resolution; ++i) {
		hipMallocManaged(&(DF->field[i]), resolution*sizeof(float*));
		hipMallocManaged(&(NF->field[i]), resolution*sizeof(int*));

		for (int j = 0; j < resolution; ++j)
		{
			hipMallocManaged(&(DF->field[i][j]), resolution*sizeof(float));
			hipMallocManaged(&(NF->field[i][j]), resolution*sizeof(int));
		}
	}

	DF->res = resolution;
	NF->res = resolution;

	// create points in the field
	PointField PF;
	genPoints( PF );

	cout << PF.field[3][13][9].loc[2] << "\n";


	//GPU code
	// setup the execution parameters:
	dim3 grid( resolution, 1, 1 );
	dim3 threads( resolution, 1, 1 );

	DF->field[32][32][64] = 15;

	for (int i = 0; i < 32; ++i)
	{
		// cout << DF->field[32][32][64+i] << "\n";
	}


	calcDistance<<< grid, threads >>> (PF, DF); 

	hipError_t errSync  = hipGetLastError();
hipError_t errAsync = hipDeviceSynchronize();
if (errSync != hipSuccess) 
  printf("\nSync kernel error: %s\n", hipGetErrorString(errSync));
if (errAsync != hipSuccess)
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	hipDeviceSynchronize();

	cout << "complete calc distance\n";

	for (int i = 0; i < 32; ++i)
	{
		cout << DF->field[32][32][64+i] << "\n";
	}




	//free point field
	for (int i = 0; i < PF.res; ++i)
	{
		for (int j = 0; j < PF.res; ++j)
		{
			for (int k = 0; k < PF.res; ++k)
			{
				hipFree(PF.field[i][j][k].loc);
			}
			hipFree(PF.field[i][j]);
		}
		hipFree(PF.field[i]);
	}
	hipFree(PF.field);


	//free the 3D fields
	for (int i = 0; i < resolution; ++i) {
		for (int j = 0; j < resolution; ++j)
		{
			hipFree(DF->field[i][j]);
			hipFree(NF->field[i][j]);
		}
		hipFree(DF->field[i]);
		hipFree(NF->field[i]);
	}
	hipFree(DF->field);
	hipFree(NF->field);
}

//Point generation functions
void printCoord(float x, float y, float z, float CS, PointField PF) {
	float X = x+genFloat(CS);
	float Y = y+genFloat(CS);
	float Z = z+genFloat(CS);

	int cx = int(x/CS);
	int cy = int(y/CS);
	int cz = int(z/CS);

	cout << X << " " << Y << " " << Z << "; " << cx << " " << cy << " " << cz << ";\n";

	PF.field[cx][cy][cz].loc[0] = X;
	PF.field[cx][cy][cz].loc[1] = Y;
	PF.field[cx][cy][cz].loc[2] = Z;

	// cout << "\t{ " << X << ", " << Y << ", " << Z << " }, ";
}

float genFloat( float max ) {
	float num = float(rand())/1000000. * 4.53632792;
	//cout << num << " ";
	return fmod(num, max);
}

void genPoints( PointField &PF ) {
	srand(time(0)*394852.);

	float cubeSize = 4.;

	float start = 0.;
	float end = 64.;

	int res = int((end-start)/cubeSize);
	PF.res = res;
	PF.CS = cubeSize;


	// PF.field = new point**[res];
	hipMallocManaged(&(PF.field), res*sizeof(point**));
	for (float i = start; i < end; i+=cubeSize)
	{
		// PF.field[(int)(i/cubeSize)] = new point*[res];
		hipMallocManaged(&(PF.field[(int)(i/cubeSize)]), res*sizeof(point*));
		for (float j = start; j < end; j+=cubeSize)
		{
			// PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)] = new point[res];
			hipMallocManaged(&(PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)]), res*sizeof(point));
			for (float k = start; k < end; k+=cubeSize)
			{
				// PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)][(int)(k/cubeSize)].loc = new float[3];
				hipMallocManaged(&(PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)][(int)(k/cubeSize)].loc), 3*sizeof(float));
				printCoord( i, j, k, cubeSize, PF);
			}
		}
	}
}

//Noise generation functions
int distVal( float dist ){
	dist = 255.*(1./dist);
	int final = (int)(dist);
	return final;
}

__device__ void getPointfromCoord( coord ID, PointField PF, point &p ) {
	int offsetX = 0;
	int offsetY = 0;
	int offsetZ = 0;

	// find offsets and tile point field
	
	if (ID.x < 0) {
		offsetX = ID.x;
		ID.x = PF.res + offsetX;
	}else if (ID.x >= PF.res) {
		offsetX = ID.x - PF.res + 1;
		ID.x = offsetX - 1;
	}
	if (ID.y < 0) {
		offsetY = ID.y;
		ID.y = PF.res + offsetY;
	}else if (ID.y >= PF.res) {
		offsetY = ID.y - PF.res + 1;
		ID.y = offsetY - 1;
	}
	if (ID.z < 0) {
		offsetZ = ID.z;
		ID.z = PF.res + offsetZ;
	}else if (ID.z >= PF.res) {
		offsetZ = ID.z - PF.res + 1;
		ID.z = offsetZ - 1;
	}

	//printf("actually access: %d, %d, %d", ID.x, ID.y, ID.z);

	//grab new points
	p.loc[0] = PF.field[ID.x][ID.y][ID.z].loc[0] + ( (float)(offsetX) * PF.CS );
	p.loc[1] = PF.field[ID.x][ID.y][ID.z].loc[1] + ( (float)(offsetY) * PF.CS );
	p.loc[2] = PF.field[ID.x][ID.y][ID.z].loc[2] + ( (float)(offsetZ) * PF.CS );

	if (offsetX < 0) {
		p.loc[0] = PF.field[ID.x][ID.y][ID.z].loc[0] - ((float)(PF.res)*PF.CS);
	}else if (offsetX >= 0) {
		p.loc[0] = PF.field[ID.x][ID.y][ID.z].loc[0] + ((float)(PF.res)*PF.CS);
	}else{
		p.loc[0] = PF.field[ID.x][ID.y][ID.z].loc[0];
	}
	if (offsetY < 0) {
		p.loc[1] = PF.field[ID.x][ID.y][ID.z].loc[1] - ((float)(PF.res)*PF.CS);
	}else if (offsetY >= 0) {
		p.loc[1] = PF.field[ID.x][ID.y][ID.z].loc[1] + ((float)(PF.res)*PF.CS);
	}else{
		p.loc[1] = PF.field[ID.x][ID.y][ID.z].loc[1];
	}
	if (offsetZ < 0) {
		p.loc[2] = PF.field[ID.x][ID.y][ID.z].loc[2] - ((float)(PF.res)*PF.CS);
	}else if (offsetZ >= 0) {
		p.loc[2] = PF.field[ID.x][ID.y][ID.z].loc[2] + ((float)(PF.res)*PF.CS);
	}else{
		p.loc[2] = PF.field[ID.x][ID.y][ID.z].loc[2];
	}

}


// (__global__ & __device__) functions to be run by GPU
// get distance from points to each point in the distance field
__global__ void calcDistance( PointField PF, DistanceField* DF ) {
	float x = 0.;
	int xCubeID = 0;
	float y = 0.;
	int yCubeID = 0;
	float z = 0.;
	int zCubeID = 0;

	// distance calc variables
	float distance = 2. * PF.CS;
	float mindistance = distance;

	point p;
	coord id;
	p.loc = new float[3];



	// point location
	// DF.field[blockID][threadID][idx]
	// X = (x coord of distance field / distance field res) * point field res * cube size

	for (int idx = 0; idx < DF->res; ++idx)
	{
		distance = 2. * PF.CS;
		mindistance = distance;

		x = ((float)(blockIdx.x) / (float)(DF->res)) * (float)(PF.res) * PF.CS;
		xCubeID = (int)( x/PF.CS );
		y = ((float)(threadIdx.x) / (float)(DF->res)) * (float)(PF.res) * PF.CS;
		yCubeID = (int)( y/PF.CS );
		z = ((float)(idx) / (float)(DF->res)) * (float)(PF.res) * PF.CS;
		zCubeID = (int)( z/PF.CS );

		//find min distance to local points in a 5x5x5 cube of points sectors
		for (int i = -2; i <= 2; ++i)
		{
			for (int j = -2; j <= 2; ++j)
			{
				for (int k = -2; k <= 2; ++k)
				{
					id.x = i + xCubeID;
					id.y = j + yCubeID;
					id.z = k + zCubeID;
					//printf("attemptting to access: %d, %d, %d -- ", id.x, id.y, id.z);
					getPointfromCoord( id, PF, p );
					distance = sqrt( pow(x-p.loc[0], 2) + pow(y-p.loc[1], 2) + pow(z-p.loc[2], 2));
					//printf("Distance: %f\n", distance);
					if ( distance < mindistance)
						mindistance = distance;
				}
			}
		}

		DF->field[blockIdx.x][threadIdx.x][idx] = distance;
	}

	delete p.loc;
}

// turn distance values into noise values
__global__ void calcNoise( DistanceField DF, NoiseField* NF ) {

}
