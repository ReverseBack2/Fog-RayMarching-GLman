#include <iostream>
#include <stdio.h>
#include <string>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <bits/stdc++.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


using namespace std;
///////////////////////////////////////////
// STRUCTS
///////////////////////////////////////////

//3D array of distances to points
typedef struct {
	int res;
	float*** field;
} DistanceField;

//3D array of int noise values
typedef struct {
	int res;
	float*** field;
} NoiseField;

//coord to keep track of array indexes
typedef struct {
	int x;
	int y;
	int z;
} coord;

//coord but in floats for points to calc
typedef struct {
	float* loc;
} point;

//3D array of points for noise field
typedef struct {
	int res;		// how many points in one direction
	float CS;		// side length of each point section
	point*** field;
} PointField;

//Point generation code
void printCoord(float x, float y, float z, float CS, PointField PF);
float genFloat( float max );
void genPoints ( PointField &PF );

//Noise generation code
int distVal( float dist );
__device__ void getPointfromCoord( coord ID, PointField PF, point &p );
__global__ void calcDistance( PointField PF, DistanceField* DF );
__global__ void calcNoise( DistanceField DF, NoiseField* NF );



///////////////////////////////////////////
// Main Program
///////////////////////////////////////////

int main() {
	cout << "starting\n";

	int resolution = 256;
	DistanceField* DF;
	NoiseField* NF;

	hipMallocManaged(&DF, sizeof(DistanceField));
	hipMallocManaged(&NF, sizeof(NoiseField));

	//allocate 3D field of distance and noise
	hipMallocManaged(&(DF->field), resolution*sizeof(float**));
	hipMallocManaged(&(NF->field), resolution*sizeof(float**));

	for (int i = 0; i < resolution; ++i) {
		hipMallocManaged(&(DF->field[i]), resolution*sizeof(float*));
		hipMallocManaged(&(NF->field[i]), resolution*sizeof(float*));

		for (int j = 0; j < resolution; ++j)
		{
			hipMallocManaged(&(DF->field[i][j]), resolution*sizeof(float));
			hipMallocManaged(&(NF->field[i][j]), resolution*sizeof(float));
		}
	}

	DF->res = resolution;
	NF->res = resolution;

	// create points in the field
	PointField PF;
	genPoints( PF );

	// cout << PF.field[3][13][9].loc[2] << "\n";


	//GPU code
	// setup the execution parameters:
	dim3 grid( resolution, 1, 1 );
	dim3 threads( resolution, 1, 1 );

	DF->field[32][32][64] = 15;

	// for (int i = 0; i < 32; ++i)
	// {
	// 	cout << DF->field[32][32][64+i] << "\n";
	// }

	cout << "starting distance Calc\n";

	calcDistance<<< grid, threads >>> (PF, DF); 
	hipDeviceSynchronize();

	cout << "finsihed distance, starting noise value calc\n";

	calcNoise<<< grid, threads >>> (*DF, NF);
	hipDeviceSynchronize();

	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) 
	  printf("\nSync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
	  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	hipDeviceSynchronize();

	cout << "complete calc noise\n";

	// for (int i = 0; i < 32; ++i)
	// {
	// 	cout << DF->field[32][32][64+i] << "\n";
	// }

	// Find Stats about distance field

	float avg, max, min, count, current;
	min = 99999999.;
	max = -1;
	count = 0;

	for (int i = 0; i < 256; ++i)
	{
		for (int j = 0; j < 256; ++j)
		{
			for (int k = 0; k < 256; ++k)
			{
				current  = DF->field[i][j][k];

				avg += current;
				count++;

				if(max < current)
					max = current;
				if(min > current)
					min = current;
			}
		}
		// cout << DF->field[32][32][64+i] << "\n";
	}

	avg = avg / count;
	cout << "min :" << min << "\nmax :" << max << "\navg :" << avg << "\n";

	// Find Stats about noise field

	// float avg, max, min, count, current;
	min = 99999999.;
	max = -1;
	count = 0;

	for (int i = 0; i < 256; ++i)
	{
		for (int j = 0; j < 256; ++j)
		{
			for (int k = 0; k < 256; ++k)
			{
				current  = NF->field[i][j][k];

				avg += current;
				count++;

				if(max < current)
					max = current;
				if(min > current)
					min = current;
			}
		}
	}

	avg = avg / count;
	cout << "min :" << min << "\nmax :" << max << "\navg :" << avg << "\n";


	// write noise texture

	FILE *fp = fopen( "noise.tex", "wb" );
	if( fp == NULL )
	{
		cout << "error opening output file\n";
	}

	int clipping = 32;

	int num = NF->res - clipping;
	int numS = num - clipping;

	

	fwrite( &numS, 4, 1, fp );
	fwrite( &numS, 4, 1, fp );
	fwrite( &numS, 4, 1, fp );

	for( int p = clipping; p < num; p++ )
	{
		for( int t = clipping; t < num; t++ )
		{
			for( int s = clipping; s < num; s++ )
			{
				float red, green, blue, alpha;

				red = NF->field[p][t][s];
				green  = 0.5;
				blue = 0.5;
				alpha = 0.5;

				fwrite( &red, 4, 1, fp );
				fwrite( &green, 4, 1, fp );
				fwrite( &blue, 4, 1, fp );
				fwrite( &alpha, 4, 1, fp );
			}
		}
	}


	//free point field
	for (int i = 0; i < PF.res; ++i)
	{
		for (int j = 0; j < PF.res; ++j)
		{
			for (int k = 0; k < PF.res; ++k)
			{
				hipFree(PF.field[i][j][k].loc);
			}
			hipFree(PF.field[i][j]);
		}
		hipFree(PF.field[i]);
	}
	hipFree(PF.field);


	//free the 3D fields
	for (int i = 0; i < resolution; ++i) {
		for (int j = 0; j < resolution; ++j)
		{
			hipFree(DF->field[i][j]);
			hipFree(NF->field[i][j]);
		}
		hipFree(DF->field[i]);
		hipFree(NF->field[i]);
	}
	hipFree(DF->field);
	hipFree(NF->field);
}

//Point generation functions
void printCoord(float x, float y, float z, float CS, PointField PF) {
	float X = x+genFloat(CS);
	float Y = y+genFloat(CS);
	float Z = z+genFloat(CS);

	int cx = int(x/CS);
	int cy = int(y/CS);
	int cz = int(z/CS);

	//cout << X << " " << Y << " " << Z << "; " << cx << " " << cy << " " << cz << ";\n";

	PF.field[cx][cy][cz].loc[0] = X;
	PF.field[cx][cy][cz].loc[1] = Y;
	PF.field[cx][cy][cz].loc[2] = Z;

	// cout << "\t{ " << X << ", " << Y << ", " << Z << " }, ";
}

float genFloat( float max ) {
	float num = float(rand())/1000000. * 4.53632792;
	//cout << num << " ";
	return fmod(num, max);
}

void genPoints( PointField &PF ) {
	srand(time(0)*394852.);

	float cubeSize = 4.;

	float start = 0.;
	float end = 64.;

	int res = int((end-start)/cubeSize);
	PF.res = res;
	PF.CS = cubeSize;


	// PF.field = new point**[res];
	hipMallocManaged(&(PF.field), res*sizeof(point**));
	for (float i = start; i < end; i+=cubeSize)
	{
		// PF.field[(int)(i/cubeSize)] = new point*[res];
		hipMallocManaged(&(PF.field[(int)(i/cubeSize)]), res*sizeof(point*));
		for (float j = start; j < end; j+=cubeSize)
		{
			// PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)] = new point[res];
			hipMallocManaged(&(PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)]), res*sizeof(point));
			for (float k = start; k < end; k+=cubeSize)
			{
				// PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)][(int)(k/cubeSize)].loc = new float[3];
				hipMallocManaged(&(PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)][(int)(k/cubeSize)].loc), 3*sizeof(float));
				printCoord( i, j, k, cubeSize, PF);
			}
		}
	}
}

//Noise generation functions
int distVal( float dist ){
	dist = 255.*(1./dist);
	int final = (int)(dist);
	return final;
}

__device__ void getPointfromCoord( coord ID, PointField PF, point &p ) {
	int offsetX = 0;
	int offsetY = 0;
	int offsetZ = 0;

	// find offsets and tile point field
	
	if (ID.x < 0) {
		offsetX = ID.x;
		ID.x = PF.res + offsetX;
	}else if (ID.x >= PF.res) {
		offsetX = ID.x - PF.res + 1;
		ID.x = offsetX - 1;
	}
	if (ID.y < 0) {
		offsetY = ID.y;
		ID.y = PF.res + offsetY;
	}else if (ID.y >= PF.res) {
		offsetY = ID.y - PF.res + 1;
		ID.y = offsetY - 1;
	}
	if (ID.z < 0) {
		offsetZ = ID.z;
		ID.z = PF.res + offsetZ;
	}else if (ID.z >= PF.res) {
		offsetZ = ID.z - PF.res + 1;
		ID.z = offsetZ - 1;
	}

	//printf("actually access: %d, %d, %d", ID.x, ID.y, ID.z);

	//grab new points
	p.loc[0] = PF.field[ID.x][ID.y][ID.z].loc[0] + ( (float)(offsetX) * PF.CS );
	p.loc[1] = PF.field[ID.x][ID.y][ID.z].loc[1] + ( (float)(offsetY) * PF.CS );
	p.loc[2] = PF.field[ID.x][ID.y][ID.z].loc[2] + ( (float)(offsetZ) * PF.CS );

	if (offsetX < 0) {
		p.loc[0] = PF.field[ID.x][ID.y][ID.z].loc[0] - ((float)(PF.res)*PF.CS);
	}else if (offsetX >= 0) {
		p.loc[0] = PF.field[ID.x][ID.y][ID.z].loc[0] + ((float)(PF.res)*PF.CS);
	}else{
		p.loc[0] = PF.field[ID.x][ID.y][ID.z].loc[0];
	}
	if (offsetY < 0) {
		p.loc[1] = PF.field[ID.x][ID.y][ID.z].loc[1] - ((float)(PF.res)*PF.CS);
	}else if (offsetY >= 0) {
		p.loc[1] = PF.field[ID.x][ID.y][ID.z].loc[1] + ((float)(PF.res)*PF.CS);
	}else{
		p.loc[1] = PF.field[ID.x][ID.y][ID.z].loc[1];
	}
	if (offsetZ < 0) {
		p.loc[2] = PF.field[ID.x][ID.y][ID.z].loc[2] - ((float)(PF.res)*PF.CS);
	}else if (offsetZ >= 0) {
		p.loc[2] = PF.field[ID.x][ID.y][ID.z].loc[2] + ((float)(PF.res)*PF.CS);
	}else{
		p.loc[2] = PF.field[ID.x][ID.y][ID.z].loc[2];
	}

}


// (__global__ & __device__) functions to be run by GPU
// get distance from points to each point in the distance field
__global__ void calcDistance( PointField PF, DistanceField* DF ) {
	float x = 0.;
	int xCubeID = 0;
	float y = 0.;
	int yCubeID = 0;
	float z = 0.;
	int zCubeID = 0;

	// distance calc variables
	float distance = 2. * PF.CS;
	float mindistance = distance;

	point p;
	coord id;
	p.loc = new float[3];



	// point location
	// DF.field[blockID][threadID][idx]
	// X = (x coord of distance field / distance field res) * point field res * cube size

	for (int idx = 0; idx < DF->res; ++idx)
	{
		distance = 2. * PF.CS;
		mindistance = distance;

		x = ((float)(blockIdx.x) / (float)(DF->res)) * (float)(PF.res) * PF.CS;
		xCubeID = (int)( x/PF.CS );
		y = ((float)(threadIdx.x) / (float)(DF->res)) * (float)(PF.res) * PF.CS;
		yCubeID = (int)( y/PF.CS );
		z = ((float)(idx) / (float)(DF->res)) * (float)(PF.res) * PF.CS;
		zCubeID = (int)( z/PF.CS );

		//find min distance to local points in a 5x5x5 cube of points sectors
		for (int i = -2; i <= 2; ++i)
		{
			for (int j = -2; j <= 2; ++j)
			{
				for (int k = -2; k <= 2; ++k)
				{
					id.x = i + xCubeID;
					id.y = j + yCubeID;
					id.z = k + zCubeID;
					//printf("attemptting to access: %d, %d, %d -- ", id.x, id.y, id.z);
					getPointfromCoord( id, PF, p );
					// distance = pow( pow(x-p.loc[0], 2) + pow(y-p.loc[1], 2) + pow(z-p.loc[2], 2), 0.5);
					distance = pow(x-p.loc[0], 2) + pow(y-p.loc[1], 2) + pow(z-p.loc[2], 2);
					//printf("Distance: %f\n", distance);
					if ( distance < mindistance)
						mindistance = distance;
				}
			}
		}

		DF->field[blockIdx.x][threadIdx.x][idx] = distance;
	}

	delete p.loc;
}

// turn distance values into noise values
__global__ void calcNoise( DistanceField DF, NoiseField* NF ) {

	// point location
	// DF.field[blockID][threadID][idx]
	// X = (x coord of distance field / distance field res) * point field res * cube size

	float value = 0;

	float max = 18000;
	float min = 64;

	for (int idx = 0; idx < DF.res; ++idx)
	{
		value = DF.field[blockIdx.x][threadIdx.x][idx];

		value = (value - min) / (max-min); 	// (~7.0->132.0) => (~0.0->1.0)
		value = 1. - value;					// (~0.0->1.0) => (1.0->0.0)

		NF->field[blockIdx.x][threadIdx.x][idx] = value;
	}
}
