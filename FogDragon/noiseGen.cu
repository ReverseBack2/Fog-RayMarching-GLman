
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <bits/stdc++.h>
#include <math.h>

using namespace std;
///////////////////////////////////////////
// STRUCTS
///////////////////////////////////////////

//3D array of distances to points
typedef struct {
	int res;
	float*** field;
} DistanceField;

//3D array of int noise values
typedef struct {
	int res;
	int*** field;
} NoiseField;

//coord to keep track of array indexes
typedef struct {
	int x;
	int y;
	int z;
} coord;

//coord but in floats for points to calc
typedef struct {
	float* loc;
} point;

//3D array of points for noise field
typedef struct {
	int res;
	point*** field;
} PointField;

//Point generation code
void printCoord(float x, float y, float z, float CS, PointField PF);
float genFloat( float max );
void genPoints ( PointField &PF );

//Noise generation code
int distVal( float dist );
coord correctPoint( coord ID, int res );
__global__ void calcNoise();


int main() {
	cout << "compiles\n";

	float resolution = 256;
	DistanceField DF;
	NoiseField NF;

	//allocate 3D field of distance and noise
	hipMallocManaged(&DF.field, resolution*sizeof(float**));
	hipMallocManaged(&NF.field, resolution*sizeof(int**));

	for (int i = 0; i < resolution; ++i) {
		hipMallocManaged(&DF.field[i], resolution*sizeof(float*));
		hipMallocManaged(&NF.field[i], resolution*sizeof(int*));

		for (int j = 0; j < resolution; ++j)
		{
			hipMallocManaged(&DF.field[i][j], resolution*sizeof(float));
			hipMallocManaged(&NF.field[i][j], resolution*sizeof(int));
		}
	}


	// create points in the field
	PointField PF;
	genPoints( PF );

	cout << PF.field[3][13][9].loc[2] << "\n";



	//free the 3D fields
	for (int i = 0; i < resolution; ++i) {
		for (int j = 0; j < resolution; ++j)
		{
			hipFree(DF.field[i][j]);
			hipFree(NF.field[i][j]);
		}
		hipFree(DF.field[i]);
		hipFree(NF.field[i]);
	}
	hipFree(DF.field);
	hipFree(NF.field);
}

//Point generation functions
void printCoord(float x, float y, float z, float CS, PointField PF) {
	float X = x+genFloat(CS);
	float Y = y+genFloat(CS);
	float Z = z+genFloat(CS);

	int cx = int(x/CS);
	int cy = int(y/CS);
	int cz = int(z/CS);

	cout << X << " " << Y << " " << Z << "; " << cx << " " << cy << " " << cz << ";\n";

	PF.field[cx][cy][cz].loc[0] = X;
	PF.field[cx][cy][cz].loc[1] = Y;
	PF.field[cx][cy][cz].loc[2] = Z;

	// cout << "\t{ " << X << ", " << Y << ", " << Z << " }, ";
}

float genFloat( float max ) {
	float num = float(rand())/1000000. * 4.53632792;
	//cout << num << " ";
	return fmod(num, max);
}

void genPoints( PointField &PF ) {
	srand(time(0)*394852.);

	float cubeSize = 4.;

	float start = 0.;
	float end = 64.;

	int res = int((end-start)/cubeSize);
	PF.res = res;


	PF.field = new point**[res];
	for (float i = start; i < end; i+=cubeSize)
	{
		PF.field[(int)(i/cubeSize)] = new point*[res];
		for (float j = start; j < end; j+=cubeSize)
		{
			PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)] = new point[res];
			for (float k = start; k < end; k+=cubeSize)
			{
				PF.field[(int)(i/cubeSize)][(int)(j/cubeSize)][(int)(k/cubeSize)].loc = new float[3];
				printCoord( i, j, k, cubeSize, PF);
			}
		}
	}

		
}

//Noise generation functions
int distVal( float dist ){
	dist = 255.*(1./dist);
	int final = (int)(dist);
	return final;
}

coord correctPoint( coord ID, int res ) {
	if (ID.x >= res)
		ID.x = 0;
	if (ID.y >= res)
		ID.y = 0;
	if (ID.z >= res)
		ID.z = 0;
	return ID;
}

// (__global__) functions to be run by GPU
__global__ void calcNoise(){


}